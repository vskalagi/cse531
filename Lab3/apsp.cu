#include "hip/hip_runtime.h"


#include <cstdio>
#include <cstdlib>
//#include "cuda_apsp.cuh"
#define BLOCK_SIZE 16
#define INF 200

/**
 * CUDA handle error, if error occurs print message and exit program
*
* @param error: CUDA error status
*/
#define HANDLE_ERROR(error) { \
    if (error != hipSuccess) { \
        fprintf(stderr, "%s in %s at line %d\n", \
                hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} \

/**
 * Naive CUDA kernel implementation algorithm Floyd Wharshall for APSP
 * check if path from vertex x -> y will be short using vertex u x -> u -> y
 * for all vertices in graph
 *
 * @param u: Index of vertex u
 * @param nvertex: Number of all vertex in graph
 * @param pitch: Length of row in memory
 * @param graph: Array of graph with distance between vertex on device
 * @param pred: Array of predecessors for a graph on device
 */
static __global__
void _naive_fw_kernel(const int u, size_t pitch, const int nvertex, int* const graph) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (y < nvertex && x < nvertex) {
        int indexYX = y * pitch + x;
        int indexUX = u * pitch + x;

        int newPath = graph[y * pitch + u] + graph[indexUX];
        int oldPath = graph[indexYX];
        if (oldPath > newPath) {
            graph[indexYX] = newPath;
        }
    }
}



/**
 * Allocate memory on device and copy memory from host to device
 * @param dataHost: Reference to unique ptr to graph data with allocated fields on host
 * @param graphDevice: Pointer to array of graph with distance between vertex on device
 * @param predDevice: Pointer to array of predecessors for a graph on device
 *
 * @return: Pitch for allocation
 */
static
size_t _cudaMoveMemoryToDevice(const int*  dataHost, int **graphDevice, int nvertex) {
    size_t height = nvertex;
    size_t width = height * sizeof(int);
    size_t pitch;

    // Allocate GPU buffers for matrix of shortest paths d(G) 
    HANDLE_ERROR(hipMallocPitch(graphDevice, &pitch, width, height));

    // Copy input from host memory to GPU buffers and
    HANDLE_ERROR(hipMemcpy2D(*graphDevice, pitch,
            dataHost, width, width, height, hipMemcpyHostToDevice));

    return pitch;
}

/**
 * Copy memory from device to host and free device memory
 *
 * @param graphDevice: Array of graph with distance between vertex on device
 * @param predDevice: Array of predecessors for a graph on device
 * @param dataHost: Reference to unique ptr to graph data with allocated fields on host
 * @param pitch: Pitch for allocation
 */
static
void _cudaMoveMemoryToHost(int *graphDevice,  int* dataHost, size_t pitch,int nvertex) {
    size_t height = nvertex;
    size_t width = height * sizeof(int);

    HANDLE_ERROR(hipMemcpy2D(dataHost, width, graphDevice, pitch, width, height, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(graphDevice));
}

/**
 * Naive implementation of Floyd Warshall algorithm in CUDA
 *
 * @param dataHost: Reference to unique ptr to graph data with allocated fields on host
 */
void cudaNaiveFW(int *dataHost,int nvertex) {
    // Choose which GPU to run on, change this on a multi-GPU system.
    HANDLE_ERROR(hipSetDevice(0));
    //int nvertex = dataHost->nvertex;

    // Initialize the grid and block dimensions here
    dim3 dimGrid((nvertex - 1) / BLOCK_SIZE + 1, (nvertex - 1) / BLOCK_SIZE + 1, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    int *graphDevice, *predDevice;
    size_t pitch = _cudaMoveMemoryToDevice(dataHost, &graphDevice, nvertex);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(_naive_fw_kernel), hipFuncCachePreferL1);
    for(int vertex = 0; vertex < nvertex; ++vertex) {
        _naive_fw_kernel<<<dimGrid, dimBlock>>>(vertex, pitch / sizeof(int), nvertex, graphDevice );
    }

    // Check for any errors launching the kernel
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
    _cudaMoveMemoryToHost(graphDevice, dataHost, pitch, nvertex);
}

/**
 * Blocked implementation of Floyd Warshall algorithm in CUDA
 *
 * @param data: unique ptr to graph data with allocated fields on host
 */
// void cudaBlockedFW(const std::unique_ptr<graphAPSPTopology>& dataHost) {
//     HANDLE_ERROR(cudaSetDevice(0));
//     int nvertex = dataHost->nvertex;
//     int *graphDevice, *predDevice;
//     size_t pitch = _cudaMoveMemoryToDevice(dataHost, &graphDevice, &predDevice);

//     dim3 gridPhase1(1 ,1, 1);
//     dim3 gridPhase2((nvertex - 1) / BLOCK_SIZE + 1, 2 , 1);
//     dim3 gridPhase3((nvertex - 1) / BLOCK_SIZE + 1, (nvertex - 1) / BLOCK_SIZE + 1 , 1);
//     dim3 dimBlockSize(BLOCK_SIZE, BLOCK_SIZE, 1);

//     int numBlock = (nvertex - 1) / BLOCK_SIZE + 1;

//     for(int blockID = 0; blockID < numBlock; ++blockID) {
//         // Start dependent phase
//         _blocked_fw_dependent_ph<<<gridPhase1, dimBlockSize>>>
//                 (blockID, pitch / sizeof(int), nvertex, graphDevice, predDevice);

//         // Start partially dependent phase
//         _blocked_fw_partial_dependent_ph<<<gridPhase2, dimBlockSize>>>
//                 (blockID, pitch / sizeof(int), nvertex, graphDevice, predDevice);

//         // Start independent phase
//         _blocked_fw_independent_ph<<<gridPhase3, dimBlockSize>>>
//                 (blockID, pitch / sizeof(int), nvertex, graphDevice, predDevice);
//     }

//     // Check for any errors launching the kernel
//     HANDLE_ERROR(cudaGetLastError());
//     HANDLE_ERROR(cudaDeviceSynchronize());
//     _cudaMoveMemoryToHost(graphDevice, predDevice, dataHost, pitch);
// }


int main(int argc, char** argv) {
    int n, m, *d;
    // input
    FILE *infile = fopen(argv[1], "r");
    fscanf(infile, "%d %d", &n, &m);
    d = (int *) malloc(sizeof(int *) * n * n);
    for (int i = 0; i < n * n; ++i) d[i] = INF;
    int a, b, w;
    for (int i = 0; i < m; ++i) {
        fscanf(infile, "%d %d %d", &a, &b, &w);
        d[a * n + b] = d[b * n + a] = w;
    }
    fclose(infile);
    cudaNaiveFW(d,n);
    // ouput
    FILE *outfile = fopen(argv[2], "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            fprintf(outfile, "%d%s",
                (i == j ? 0 : d[i * n + j]),
                (j == n - 1 ? " \n" : " ")
            );
        }
    }
    free(d);
}
